#include "matmul.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>

void timeMatmul1(int n, int block_dim) {
    int *A = new int[n * n];
    int *B = new int[n * n];

    srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        A[i] = -100.0 + (200.0 * rand() / RAND_MAX);
        B[i] = -100.0 + (200.0 * rand() / RAND_MAX);
    }

    int *dA;
    int *dB;
    hipMalloc((void **)&dA, n * n * sizeof(int));
    hipMalloc((void **)&dB, n * n * sizeof(int));

    hipMemcpy(dA, A, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(int), hipMemcpyHostToDevice);

    int *dC;
    hipMalloc((void **)&dC, n * n * sizeof(int));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_1(dA, dB, dC, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    int *C = new int[n * n];
    hipMemcpy(C, dC, n * n * sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", C[0]);
    printf("%d\n", C[n * n - 1]);
    printf("%f\n", elapsedTime);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void timeMatmul2(int n, int block_dim) {
    float *A = new float[n * n];
    float *B = new float[n * n];

    srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        A[i] = -100.0 + (200.0 * rand() / RAND_MAX);
        B[i] = -100.0 + (200.0 * rand() / RAND_MAX);
    }

    float *dA;
    float *dB;
    hipMalloc((void **)&dA, n * n * sizeof(float));
    hipMalloc((void **)&dB, n * n * sizeof(float));

    hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    float *dC;
    hipMalloc((void **)&dC, n * n * sizeof(float));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_2(dA, dB, dC, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float *C = new float[n * n];
    hipMemcpy(C, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", C[0]);
    printf("%f\n", C[n * n - 1]);
    printf("%f\n", elapsedTime);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void timeMatmul3(int n, int block_dim) {
    double *A = new double[n * n];
    double *B = new double[n * n];

    srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        A[i] = -100.0 + (200.0 * rand() / RAND_MAX);
        B[i] = -100.0 + (200.0 * rand() / RAND_MAX);
    }

    double *dA;
    double *dB;
    hipMalloc((void **)&dA, n * n * sizeof(double));
    hipMalloc((void **)&dB, n * n * sizeof(double));

    hipMemcpy(dA, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(double), hipMemcpyHostToDevice);

    double *dC;
    hipMalloc((void **)&dC, n * n * sizeof(double));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_3(dA, dB, dC, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    double *C = new double[n * n];
    hipMemcpy(C, dC, n * n * sizeof(double), hipMemcpyDeviceToHost);

    printf("%f\n", C[0]);
    printf("%f\n", C[n * n - 1]);
    printf("%f\n", elapsedTime);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);
    int block_dim = atoi(argv[2]);

    timeMatmul1(n, block_dim);
    timeMatmul2(n, block_dim);
    timeMatmul3(n, block_dim);

    return 0;
}
