#include "reduce.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);

    float *input = new float[N];

    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        input[i] = -1.0 + (2.0 * rand() / RAND_MAX);
    }

    float *dInput;
    hipMalloc((void **)&dInput, N * sizeof(float));

    hipMemcpy(dInput, input, N * sizeof(float), hipMemcpyHostToDevice);

    int blockCount = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);

    float *dOutput;
    hipMalloc((void **)&dOutput, blockCount * sizeof(float));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce(&dInput, &dOutput, N, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float sum;
    hipMemcpy(&sum, dInput, sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", sum);
    printf("%f\n", elapsedTime);

    delete[] input;
    hipFree(dInput);
    hipFree(dOutput);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
