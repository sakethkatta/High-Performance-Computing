
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>

__global__ void computeLinearCombination(int *dA, int a) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    dA[(y * 8) + x] = (a * x) + y;
}

int main() {
    int *dA;
    hipMalloc((void **)&dA, 16 * sizeof(int));

    srand(time(NULL));
    int a = 20.0 * rand() / RAND_MAX;

    computeLinearCombination<<<2, 8>>>(dA, a);
    hipDeviceSynchronize();

    int *hA = new int[16];
    hipMemcpy(hA, dA, 16 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 15; i++) {
        printf("%d ", hA[i]);
    }
    printf("%d\n", hA[15]);

    hipFree(dA);
    delete[] hA;

    return 0;
}
