#include "vscale.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>

int main(int argc, char **argv) {
    int n = atoi(argv[1]);
    float *a = new float[n];
    float *b = new float[n];

    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        a[i] = -10.0 + (20.0 * rand() / RAND_MAX);
        b[i] = 0.0 + (1.0 * rand() / RAND_MAX);
    }

    float *dA;
    float *dB;
    hipMalloc((void **)&dA, n * sizeof(float));
    hipMalloc((void **)&dB, n * sizeof(float));

    hipMemcpy(dA, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, n * sizeof(float), hipMemcpyHostToDevice);

    int threads = 512;
    int blocks = (n + threads - 1) / threads;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vscale<<<blocks, threads>>>(dA, dB, n);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(b, dB, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", elapsedTime);
    printf("%f\n", b[0]);
    printf("%f\n", b[n - 1]);

    delete[] a;
    delete[] b;
    hipFree(dA);
    hipFree(dB);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
