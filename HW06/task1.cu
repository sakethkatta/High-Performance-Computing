#include "matmul.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);

    float *A = new float[n * n];
    float *B = new float[n * n];

    srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        A[i] = -1.0 + (2.0 * rand() / RAND_MAX);
        B[i] = -1.0 + (2.0 * rand() / RAND_MAX);
    }

    float *dA;
    float *dB;
    hipMalloc((void **)&dA, n * n * sizeof(float));
    hipMalloc((void **)&dB, n * n * sizeof(float));

    hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    float *dC;
    hipMalloc((void **)&dC, n * n * sizeof(float));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul(dA, dB, dC, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float *C = new float[n * n];
    hipMemcpy(C, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", C[n * n - 1]);
    printf("%f\n", elapsedTime);

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
