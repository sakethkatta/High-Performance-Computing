#include "stencil.cuh"
#include <cstdio>
#include <cstdlib>
#include <ctime>

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);

    float *image = new float[n];
    float *mask = new float[2 * R + 1];

    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        image[i] = -1.0 + (2.0 * rand() / RAND_MAX);
    }
    for (int i = 0; i <= 2 * R; i++) {
        mask[i] = -1.0 + (2.0 * rand() / RAND_MAX);
    }

    float *dImage;
    float *dMask;
    hipMalloc((void **)&dImage, n * sizeof(float));
    hipMalloc((void **)&dMask, (2 * R + 1) * sizeof(float));

    hipMemcpy(dImage, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dMask, mask, (2 * R + 1) * sizeof(float),
               hipMemcpyHostToDevice);

    float *dOutput;
    hipMalloc((void **)&dOutput, n * sizeof(float));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    stencil(dImage, dMask, dOutput, n, R, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float *output = new float[n];
    hipMemcpy(output, dOutput, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", output[n - 1]);
    printf("%f\n", elapsedTime);

    delete[] image;
    delete[] mask;
    delete[] output;
    hipFree(dImage);
    hipFree(dMask);
    hipFree(dOutput);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
